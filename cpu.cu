#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include <cmath>
#include <chrono>
using namespace std::chrono;

__global__ void cuda_hello(){
  printf("Hello World from GPU \n");
}
using namespace std;
int main () {
 cout<<"Input number of elements: ";
 int n;
 cin>>n;
 vector<int> arr1(n);
 vector<int> arr2(n);
 vector<int> res(n);
 auto start= high_resolution_clock::now();
 for(int i=0;i<n;i++){
  arr1[i] = rand()%10000+1;
  arr2[i] = rand()%10000+1;
  res[i]= arr1[i]+arr2[i];
 }
 auto stop= high_resolution_clock::now();
 auto duration = duration_cast<microseconds>(stop-start);
 cout<<"Time taken: "<< duration.count()<<endl;
 return 0;